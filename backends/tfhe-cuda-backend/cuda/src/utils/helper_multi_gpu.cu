#include "hip/hip_runtime.h"
#include "device.h"
#include "helper_multi_gpu.h"
#include <mutex>
#include <omp.h>

std::mutex m;
bool p2p_enabled = false;

int cuda_setup_multi_gpu() {
  int num_gpus = cuda_get_number_of_gpus();
  if (num_gpus == 0)
    PANIC("GPU error: the number of GPUs should be > 0.")
  int num_used_gpus = 1;
  if (num_gpus > 1) {
    m.lock();
    if (!p2p_enabled) {
      p2p_enabled = true;
      omp_set_nested(1);
      int has_peer_access_to_device_0;
      for (int i = 1; i < num_gpus; i++) {
        check_cuda_error(
            hipDeviceCanAccessPeer(&has_peer_access_to_device_0, i, 0));
        if (has_peer_access_to_device_0) {
          hipMemPool_t mempool;
          hipMemAccessDesc desc = {};
          // Enable P2P Access and mempool access
          check_cuda_error(hipSetDevice(i));
          check_cuda_error(hipDeviceEnablePeerAccess(0, 0));

          check_cuda_error(hipDeviceGetDefaultMemPool(&mempool, 0));
          desc.location.type = hipMemLocationTypeDevice;
          desc.location.id = i;
          desc.flags = hipMemAccessFlagsProtReadWrite;
          check_cuda_error(
              hipMemPoolSetAccess(mempool, &desc, 1 /* numDescs */));
          num_used_gpus += 1;
        } else {
          break;
        }
      }
    } else {
      int has_peer_access_to_device_0;
      for (int i = 1; i < num_gpus; i++) {
        check_cuda_error(
            hipDeviceCanAccessPeer(&has_peer_access_to_device_0, i, 0));
        if (has_peer_access_to_device_0) {
          num_used_gpus += 1;
        } else {
          break;
        }
      }
    }
    m.unlock();
  }
  return num_used_gpus;
}

int get_active_gpu_count(int num_inputs, int gpu_count) {
  int active_gpu_count = gpu_count;
  if (gpu_count > num_inputs) {
    active_gpu_count = num_inputs;
  }
  return active_gpu_count;
}

int get_gpu_offset(int total_num_inputs, int gpu_index, int gpu_count) {
  int gpu_offset = 0;
  for (uint i = 0; i < gpu_index; i++)
    gpu_offset += get_num_inputs_on_gpu(total_num_inputs, i, gpu_count);
  return gpu_offset;
}

int get_num_inputs_on_gpu(int total_num_inputs, int gpu_index, int gpu_count) {

  int num_inputs = 0;
  // If there are fewer inputs than GPUs, not all GPUs are active and GPU 0
  // handles everything
  if (gpu_count > total_num_inputs) {
    if (gpu_index < total_num_inputs) {
      num_inputs = 1;
    }
  } else {
    // If there are more inputs than GPUs, all GPUs are active and compute over
    // a chunk of the total inputs. The chunk size is smaller on the last GPUs.
    int small_input_num, large_input_num, cutoff;
    if (total_num_inputs % gpu_count == 0) {
      small_input_num = total_num_inputs / gpu_count;
      large_input_num = small_input_num;
      cutoff = 0;
    } else {
      int y = ceil((double)total_num_inputs / (double)gpu_count) * gpu_count -
              total_num_inputs;
      cutoff = gpu_count - y;
      small_input_num = total_num_inputs / gpu_count;
      large_input_num = (int)ceil((double)total_num_inputs / (double)gpu_count);
    }
    if (gpu_index < cutoff)
      num_inputs = large_input_num;
    else
      num_inputs = small_input_num;
  }
  return num_inputs;
}

/// Load an array residing on one GPU to all active gpus
/// and split the array among them
template <typename Torus>
void multi_gpu_dispatch(hipStream_t *streams, uint32_t *gpu_indexes,
                        uint32_t gpu_count, std::vector<Torus *> &dest,
                        Torus *src, uint32_t num_inputs, uint32_t elements_per_input) {

    auto active_gpu_count = get_active_gpu_count(num_inputs, gpu_count);
#pragma omp parallel for num_threads(active_gpu_count)
    for (uint i = 0; i < active_gpu_count; i++) {
        auto gpu_len = get_num_inputs_on_gpu(num_inputs, i, gpu_count) * elements_per_input;
        Torus *array = (Torus *)cuda_malloc_async(gpu_len * sizeof(Torus), streams[i], gpu_indexes[i]);
        check_cuda_error(cuda_memcpy_async_gpu_to_gpu(
                array, src + get_gpu_offset(num_inputs, i, gpu_count) * elements_per_input,
                gpu_len * sizeof(Torus),
                streams[i], gpu_indexes[i]));
        cuda_synchronize_stream(streams[i], gpu_indexes[i]);
        dest.push_back(array);
    }
}

/// Copy data from multiple GPUs back to GPU 0
template <typename Torus>
void multi_gpu_gather(hipStream_t *streams, uint32_t *gpu_indexes,
                      uint32_t gpu_count, Torus *dest, std::vector<Torus *> src,
                      uint32_t num_inputs, uint32_t elements_per_input) {

    auto active_gpu_count = get_active_gpu_count(num_inputs, gpu_count);
#pragma omp parallel for num_threads(active_gpu_count)
    for (uint i = 0; i < active_gpu_count; i++) {
        auto gpu_len = get_num_inputs_on_gpu(num_inputs, i, gpu_count) * elements_per_input;
        check_cuda_error(cuda_memcpy_async_gpu_to_gpu(
                dest[get_gpu_offset(num_inputs, i, gpu_count) * elements_per_input],
                src[i],
                gpu_len * sizeof(Torus),
                streams[i], gpu_indexes[i]));
        cuda_synchronize_stream(streams[i], gpu_indexes[i]);
    }
}

template <typename Torus>
void multi_gpu_release(hipStream_t *streams, uint32_t *gpu_indexes,
                       uint32_t gpu_count, std::vector<Torus *> vec,
                       uint32_t num_inputs) {

    auto active_gpu_count = get_active_gpu_count(num_inputs, gpu_count);
#pragma omp parallel for num_threads(active_gpu_count)
    for (uint i = 0; i < active_gpu_count; i++) {
        check_cuda_error(cuda_drop_asyc(vec[i], streams[i], gpu_indexes[i]));
        cuda_synchronize_stream(streams[i], gpu_indexes[i]);
    }
}
